
#include <hip/hip_runtime.h>

// CudafyExamples.Arrays.ArrayBasicIndexing
extern "C" __global__  void add_0( int* a, int aLen0,  int* b, int bLen0,  int* c, int cLen0);
// CudafyExamples.Arrays.ArrayBasicIndexing
extern "C" __global__  void add_1( int* a, int aLen0,  int* b, int bLen0,  int* c, int cLen0);
// CudafyExamples.Arrays.ArrayBasicIndexing
extern "C" __global__  void add_2( int* a, int aLen0,  int* b, int bLen0,  int* c, int cLen0);
// CudafyExamples.Arrays.ArrayBasicIndexing
extern "C" __global__  void add_3( int* a, int aLen0,  int* b, int bLen0,  int* c, int cLen0);
// CudafyExamples.Arrays.ArrayBasicIndexing
extern "C" __global__  void add_4( int* a, int aLen0,  int* b, int bLen0,  int* c, int cLen0);

// CudafyExamples.Arrays.ArrayBasicIndexing
extern "C" __global__  void add_0( int* a, int aLen0,  int* b, int bLen0,  int* c, int cLen0)
{
	for (int i = blockIdx.x; i < 1024; i += gridDim.x)
	{
		c[(i)] = a[(i)] + b[(i)];
	}
}
// CudafyExamples.Arrays.ArrayBasicIndexing
extern "C" __global__  void add_1( int* a, int aLen0,  int* b, int bLen0,  int* c, int cLen0)
{
	for (int i = blockIdx.x; i < aLen0; i += gridDim.x)
	{
		c[(i)] = a[(i)] + b[(i)];
	}
}
// CudafyExamples.Arrays.ArrayBasicIndexing
extern "C" __global__  void add_2( int* a, int aLen0,  int* b, int bLen0,  int* c, int cLen0)
{
	for (int i = blockIdx.x; i < bLen0; i += gridDim.x)
	{
		c[(i)] = a[(i)] + b[(i)];
	}
}
// CudafyExamples.Arrays.ArrayBasicIndexing
extern "C" __global__  void add_3( int* a, int aLen0,  int* b, int bLen0,  int* c, int cLen0)
{
	for (int i = blockIdx.x; i < cLen0; i += gridDim.x)
	{
		c[(i)] = a[(i)] + b[(i)];
	}
}
// CudafyExamples.Arrays.ArrayBasicIndexing
extern "C" __global__  void add_4( int* a, int aLen0,  int* b, int bLen0,  int* c, int cLen0)
{
	int i = blockIdx.x;
	int rank = 1;
	while (i < cLen0)
	{
		c[(i)] = a[(i)] + b[(i)];
		i += gridDim.x;
	}
}
