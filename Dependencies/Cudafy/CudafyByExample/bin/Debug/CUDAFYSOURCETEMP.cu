
#include <hip/hip_runtime.h>

// CudafyByExample.hist_gpu_shmem_atomics
extern "C" __global__  void histo_kernel( unsigned char* buffer, int bufferLen0, int size,  unsigned int* histo, int histoLen0);

// CudafyByExample.hist_gpu_shmem_atomics
extern "C" __global__  void histo_kernel( unsigned char* buffer, int bufferLen0, int size,  unsigned int* histo, int histoLen0)
{
	__shared__ unsigned int array[256];

	int arrayLen0 = 256;
	array[(threadIdx.x)] = 0u;
	__syncthreads();
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int num = blockDim.x * gridDim.x;
	while (i < size)
	{
		atomicAdd(&array[((int)buffer[(i)])], 1u);
		i += num;
	}
	__syncthreads();
	atomicAdd(&histo[(threadIdx.x)], array[(threadIdx.x)]);
}
